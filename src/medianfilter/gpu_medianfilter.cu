#include "hip/hip_runtime.h"
#include <memory.h>
#include <hip/hip_runtime.h>
#include <opencv2/core/core.hpp>
#include "gpu_medianfilter.h"

__global__ void _medianfilter1D(const element* signal, element* result, unsigned length, int w_width, int ts_per_bk)
{
	element *window = (element*)malloc(sizeof(element) * w_width);
    int radius = w_width / 2;
    extern __shared__ element cache[];

	int gindex = threadIdx.x + blockDim.x * blockIdx.x;
	int lindex = threadIdx.x + radius;
	// Reads input elements into shared memory
	cache[lindex] = signal[gindex];
	if (threadIdx.x < radius)
	{
		cache[lindex - radius] = signal[gindex - radius];
		cache[lindex + ts_per_bk] = signal[gindex + ts_per_bk];
	}
	__syncthreads();
	for (int j = 0; j < 2 * radius + 1; ++j)
		window[j] = cache[threadIdx.x + j];
	// Orders elements (only half of them)
	for (int j = 0; j < radius + 1; ++j)
	{
		// Finds position of minimum element
		int min = j;
		for (int k = j + 1; k < 2 * radius + 1; ++k)
			if (window[k] < window[min])
				min = k;
		// Puts found minimum element in its place
		const element temp = window[j];
		window[j] = window[min];
		window[min] = temp;
	}
	// Gets result - the middle element
	result[gindex] = window[radius];
    free(window);
}

void medianfilter1D(element* signal, element* result, unsigned length, int w_width, int ts_per_bk)
{
	element *dev_extension, *dev_result;
    int radius = w_width / 2;

	//   Check arguments
	if (!signal || length < 1)
		return;
	//   Treat special case length = 1
	if (length == 1)
	{
		if (result)
			result[0] = signal[0];
		return;
	}
	//   Allocate memory for signal extension
	element* extension = (element*)malloc((length + 2 * radius) * sizeof(element));
	//   Check memory allocation
	if (!extension)
		return;
	//   Create signal extension
	hipMemcpy(extension + 2, signal, length * sizeof(element), hipMemcpyHostToHost);
    // Marginal elements
	for (int i = 0; i < radius; ++i)
	{
		extension[i] = signal[radius - i - 1];
		extension[length + radius + i] = signal[length - 1 - i];
	}

	hipMalloc((void**)&dev_extension, (length + 2 * radius) * sizeof(element));
	hipMalloc((void**)&dev_result, length * sizeof(element));

	// Copies signal to device
	hipMemcpy(dev_extension, extension, (length + 2 * radius) * sizeof(element), hipMemcpyHostToDevice);

    // Set up execution configuration
    dim3 block(ts_per_bk, 1);
    dim3 grid((length + block.x - 1) / block.x, 1);
    unsigned shared_size = ts_per_bk + 2 * (w_width / 2);

	//   Call median filter implementation
	_medianfilter1D<<<grid, block, shared_size>>>(dev_extension + radius, dev_result, length, w_width, ts_per_bk);
	// Copies result to host
	hipMemcpy(result, dev_result, length * sizeof(element), hipMemcpyDeviceToHost);

	// Free memory
	free(extension);
	hipFree(dev_extension);
	hipFree(dev_result);
}

__global__ void _medianfilter2D(const element* signal, element* result, unsigned width, unsigned height, int k_width, int ts_per_dm)
{
	element *kernel = (element*)malloc(sizeof(element) * k_width * k_width);
    int radius = k_width / 2;
    // use dynamic size shared memory
    extern __shared__ element cache[];
    int sh_cols = ts_per_dm + radius * 2;
    int sh_rows = ts_per_dm + radius * 2;
    //int bk_cols = ts_per_dm;    int bk_rows = ts_per_dm;
    unsigned sg_cols = width + radius * 2;
    unsigned sg_rows = height + radius * 2;

	int gl_ix = threadIdx.x + blockDim.x * blockIdx.x;
	int gl_iy = threadIdx.y + blockDim.y * blockIdx.y;
    int ll_ix = threadIdx.x + radius;
    int ll_iy = threadIdx.y + radius;

	// Reads input elements into shared memory
	cache[ll_iy * sh_cols + ll_ix] = signal[gl_iy * sg_cols + gl_ix];
	if (threadIdx.x < radius)
	{
        for (int i = 0; i < radius; i++)
        {
            cache[ll_iy * sh_cols + ll_ix - radius + i] = signal[gl_iy * sg_cols + gl_ix + radius - 1 - i];
            cache[ll_iy * sh_cols + sh_cols + ll_ix - i - 1] = signal[gl_iy * sg_cols + sg_cols + gl_ix - radius + i];
        }
	}
	if (threadIdx.y < radius)
	{
        for (int i = 0; i < radius; i++)
        {
            cache[ll_ix * sh_rows + ll_iy - radius + i] = signal[gl_ix * sg_rows + gl_iy + radius - 1 - i];
            cache[ll_ix * sh_rows + sh_rows + ll_iy - i - 1] = signal[gl_ix * sg_rows + sg_rows + gl_iy - radius + i];
        }
	}
	__syncthreads();

    // Get kernel element 
    for (int i = 0; i < k_width; ++i)
	    for (int j = 0; j < k_width * k_width; ++j)
	    	kernel[j] = cache[(threadIdx.y + i) * k_width + threadIdx.x + j];

	// Orders elements (only half of them)
	for (int j = 0; j < k_width * k_width / 2 + 1; ++j)
	{
		// Finds position of minimum element
		int min = j;
		for (int k = j + 1; k < 2 * radius + 1; ++k)
			if (kernel[k] < kernel[min])
				min = k;
		// Puts found minimum element in its place
		const element temp = kernel[j];
		kernel[j] = kernel[min];
		kernel[min] = temp;
	}
	// Gets result - the middle element
	result[gl_iy * sg_cols + gl_ix] = kernel[k_width * k_width / 2];
    free(kernel);
}

void medianfilter2D(const cv::Mat &src, cv::Mat &dst, int k_width, int ts_per_dm)
{
    unsigned width = src.size().width;
    unsigned height = src.size().height;

	//   Check arguments
	if (!src.data || width < 1 || height < 1)
		return;
    if (!src.isContinuous())
        return; 

	//   Treat special case length = 1
	if (width == 1 && height == 1)
	{
        dst = src;
		return;
	}

	element *dev_extension, *dev_result;
    element *extension, *result;
    int radius = k_width / 2;

	/////   Allocate page-locked memory for image extension 
	hipHostMalloc((void**)&extension, (width + 2 * radius) * (height + 2 * radius) * sizeof(element));
    hipHostMalloc((void**)&result, width * height * sizeof(element));
	//   Check memory allocation
	if (!extension)
		return;

	/////   Create image extension
    // Inner elements
    for (unsigned i = 0; i < height; ++i)
        hipMemcpy(extension + (width + radius + radius) * (i + radius) +  radius, src.data + width * i, width * sizeof(element), hipMemcpyHostToHost);
    // marginal elements
    for (int i = 0; i < radius; ++i)
    {
        hipMemcpy(extension + (width + radius + radius) * (radius - i - 1), src.data + width * i, width * sizeof(element), hipMemcpyHostToHost);
        hipMemcpy(extension + (width + radius + radius) * (height + radius + i), src.data + width * (height - i - 1), width * sizeof(element), hipMemcpyHostToHost); 
    }
	for (int i = 0; i < height; ++i)
	{
        for (int j = 0; j < radius; ++j)
        {
		    extension[(width + radius + radius) * (radius + i) + j] = src.data[width * i + radius - 1  - j];
		    extension[(width + radius + radius) * (radius + i) + width + radius + j] = src.data[width * i + width - (radius - j - 1)];
        }
	}

    // Allocate device memory
	hipMalloc((void**)&dev_extension, (width + 2 * radius) * (height + 2 * radius) * sizeof(element));
	hipMalloc((void**)&dev_result, width * height * sizeof(element));

	// Copies extension to device
	hipMemcpy(dev_extension, extension, (width + 2 * radius) * (height + 2 * radius) * sizeof(element), hipMemcpyHostToDevice);

    // Set up execution configuration
    dim3 block(ts_per_dm, ts_per_dm);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
    unsigned shared_size = (ts_per_dm + 2 * radius) * (ts_per_dm + 2 * radius);

	//   Call median filter implementation
	_medianfilter2D<<<grid, block, shared_size>>>(dev_extension + radius, dev_result, width, height, k_width, ts_per_dm);
	// Copies result to host
	hipMemcpy(result, dev_result, width * height * sizeof(element), hipMemcpyDeviceToHost);

    element *dstData = (element*)malloc(width * height * sizeof(element));
    hipMemcpy(dstData, result, width * height * sizeof(element), hipMemcpyHostToHost);

    // Create dst image
    dst = cv::Mat(height, width, src.type(), dstData);

	// Free memory
	hipHostFree(extension);
	hipFree(dev_extension);
	hipFree(dev_result);
}
