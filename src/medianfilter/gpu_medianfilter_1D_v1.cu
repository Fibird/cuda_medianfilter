#include "hip/hip_runtime.h"
/**
 * Author: Liu Chaoyang
 * E-mail: chaoyanglius@gmail.com
 * 
 * Median filter using cuda C(global memory)
 * Copyright (C) 2018 Liu Chaoyang
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <stdio.h>
#include <memory.h>
#include "waveformat/waveformat.h"
#include <hip/hip_runtime.h>

#define WINDOW_WIDTH 5

// signal/image element type
typedef short element;

//   1D MEDIAN FILTER implementation
//     signal - input signal
//     result - output signal
//     length - length of the signal
__global__ void _medianfilter(const element* signal, element* result, int length)
{
	element window[WINDOW_WIDTH];
	int gindex = threadIdx.x + blockDim.x * blockIdx.x;
	int radius = WINDOW_WIDTH / 2;

	while (gindex < length)
	{
		for (int j = 0; j < 2 * radius + 1; ++j)
			window[j] = signal[gindex + j];
		// Orders elements (only half of them)
		for (int j = 0; j < radius + 1; ++j)
		{
			// Finds position of minimum element
			int min = j;
			for (int k = j + 1; k < 2 * radius + 1; ++k)
				if (window[k] < window[min])
					min = k;
			// Puts found minimum element in its place
			const element temp = window[j];
			window[j] = window[min];
			window[min] = temp;
		}
		// Gets result - the middle element
		result[gindex] = window[radius];
		gindex += blockDim.x * gridDim.x; 
	}
}

//   1D MEDIAN FILTER wrapper
//     signal - input signal
//     result - output signal
//     length - length of the signal
void medianfilter(element* signal, element* result, int length)
{
	element *dev_extension, *dev_result;
    int radius = WINDOW_WIDTH / 2;

	//   Check arguments
	if (!signal || length < 1)
		return;
	//   Treat special case N = 1
	if (length == 1)
	{
		if (result)
			result[0] = signal[0];
		return;
	}
	//   Allocate memory for signal extension
	element* extension = (element*)malloc((length + 2 * radius) * sizeof(element));
	//   Check memory allocation
	if (!extension)
		return;
	//   Create signal extension
	hipMemcpy(extension + 2, signal, length * sizeof(element), hipMemcpyHostToHost);
	for (int i = 0; i < radius; ++i)
	{
		extension[i] = signal[1 - i];
		extension[length + radius + i] = signal[length - 1 - i];
	}

	hipMalloc((void**)&dev_extension, (length + 2 * radius) * sizeof(int));
	hipMalloc((void**)&dev_result, length * sizeof(int));

	// Copies signal to device
	hipMemcpy(dev_extension, extension, (length + 2 * radius) * sizeof(element), hipMemcpyHostToDevice);
    
    // Set up execution configuration
    dim3 block(1024, 1);
    dim3 grid((length + block.x - 1) / block.x, 1);

	//   Call median filter implementation
	_medianfilter<<<grid, block>>>(dev_extension, dev_result, length);

	// Copies result to host
	hipMemcpy(result, dev_result, length * sizeof(element), hipMemcpyDeviceToHost);

	// Free memory
	free(extension);
	hipFree(dev_extension);
	hipFree(dev_result);
}

int main(int argc, char **argv)
{
	element *signal, *result;
    
    if (argc != 2)
    {
        printf("Please specify name of file!\n");
        exit(EXIT_FAILURE);
    }

	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

    // read input music file
	FILE *fp;
    fp = fopen(argv[1], "rb");
    if (!fp)
    {
        printf("open input file failed!\n");
        return -1;
    }
    // get header info of input file
    waveFormat fmt = readWaveHeader(fp);
    int size = fmt.data_size;
    
    // allocate host memory for input and output data
    signal = (element *)malloc(size * sizeof(element));
    // move fp to the beginning position of data
    fseek(fp, 44L, SEEK_SET);     
    // read signal data from input file
    fread(signal, sizeof(short), size, fp);
    // close file stream
    if (fp)
    {
        fclose(fp);
        fp = NULL;
    }

    // allocate host memory for output data
	result = (element *)malloc(size * sizeof(element));

    // execute median filter and time it
	hipEventRecord(start, 0);
	medianfilter(signal, result, size);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("%.3lf ms\n", elapsedTime);

    // save output data
	fp = fopen("audios/gpu_v1_rst.wav", "wb+");

	if (fp == NULL)
        printf("Open output file failed!\n");

    // write header info into output file
    writeWaveHeader(fmt, fp);
    // move fp to the beginning position of data
    fseek(fp, 44L, SEEK_SET);
    // write result data into output file
    fwrite(result, sizeof(short), size, fp);
    
    // close file stream
    if (fp)
    {
	    fclose(fp);
        fp = NULL;
    }

    // free host memory
    free(signal);
    free(result);

	return 0;
}
