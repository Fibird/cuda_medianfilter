#include <stdio.h>
#include "gpu_medianfilter.h"
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

using namespace cv;

int main(int argc, char **argv)
{
    if (argc != 2)
    {
        printf("Please specify name of file!\n");
        exit(EXIT_FAILURE);
    }
    Mat image = imread(argv[1], CV_LOAD_IMAGE_GRAYSCALE);
    image.convertTo(image, CV_32FC1);
    Mat result;
    
	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
    hipEventCreate(&stop);

    // execute median filter and time it
	hipEventRecord(start, 0);

	medianfilter2D(image, result, 3, 32);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("%.3lf ms\n", elapsedTime);
    result.convertTo(result, CV_8UC1);
    imwrite("result/restor.jpg", result);
	return 0;
}

