#include "hip/hip_runtime.h"
/**
 * Author: Liu Chaoyang
 * E-mail: chaoyanglius@gmail.com
 * 
 * Median filter using cuda C(shared memory)
 * Copyright (C) 2018 Liu Chaoyang
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <stdio.h>
#include <memory.h>
#include <hip/hip_runtime.h>
#include "waveformat/waveformat.h"

#define WINDOW_WIDTH 9
#define THREADS_PER_BLOCK 1024

// Signal/image element type
typedef int element;
//   1D MEDIAN FILTER implementation
//     signal - input signal
//     result - output signal
//     length - length of the signal
__global__ void _medianfilter(const element* signal, element* result, int length)
{
	element window[WINDOW_WIDTH];
    int radius = WINDOW_WIDTH / 2;
	__shared__ element cache[THREADS_PER_BLOCK + 2 * (WINDOW_WIDTH / 2)];

	int gindex = threadIdx.x + blockDim.x * blockIdx.x;
	int lindex = threadIdx.x + radius;
	// Reads input elements into shared memory
	cache[lindex] = signal[gindex];
	if (threadIdx.x < radius)
	{
		cache[lindex - radius] = signal[gindex - radius];
		cache[lindex + THREADS_PER_BLOCK] = signal[gindex + THREADS_PER_BLOCK];
	}
	__syncthreads();
	for (int j = 0; j < 2 * radius + 1; ++j)
		window[j] = cache[threadIdx.x + j];
	// Orders elements (only half of them)
	for (int j = 0; j < radius + 1; ++j)
	{
		// Finds position of minimum element
		int min = j;
		for (int k = j + 1; k < 2 * radius + 1; ++k)
			if (window[k] < window[min])
				min = k;
		// Puts found minimum element in its place
		const element temp = window[j];
		window[j] = window[min];
		window[min] = temp;
	}
	// Gets result - the middle element
	result[gindex] = window[radius];
}

//   1D MEDIAN FILTER wrapper
//     signal - input signal
//     result - output signal
//     length - length of the signal
void medianfilter(element* signal, element* result, int length)
{
	element *dev_extension, *dev_result;
    int radius = WINDOW_WIDTH / 2;

	//   Check arguments
	if (!signal || length < 1)
		return;
	//   Treat special case length = 1
	if (length == 1)
	{
		if (result)
			result[0] = signal[0];
		return;
	}
	//   Allocate memory for signal extension
	element* extension = (element*)malloc((length + 2 * radius) * sizeof(element));
	//   Check memory allocation
	if (!extension)
		return;
	//   Create signal extension
	hipMemcpy(extension + 2, signal, length * sizeof(element), hipMemcpyHostToHost);
	for (int i = 0; i < radius; ++i)
	{
		extension[i] = signal[1 - i];
		extension[length + radius + i] = signal[length - 1 - i];
	}

	hipMalloc((void**)&dev_extension, (length + 2 * radius) * sizeof(int));
	hipMalloc((void**)&dev_result, length * sizeof(int));

	// Copies signal to device
	hipMemcpy(dev_extension, extension, (length + 2 * radius) * sizeof(element), hipMemcpyHostToDevice);

    // Set up execution configuration
    dim3 block(THREADS_PER_BLOCK, 1);
    dim3 grid((length + block.x - 1) / block.x, 1);

	//   Call median filter implementation
	_medianfilter<<<grid, block>>>(dev_extension + radius, dev_result, length);
	// Copies result to host
	hipMemcpy(result, dev_result, length * sizeof(element), hipMemcpyDeviceToHost);

	// Free memory
	free(extension);
	hipFree(dev_extension);
	hipFree(dev_result);
}

int main(int argc, char **argv)
{
	element *signal, *result;
    
    if (argc != 2)
    {
        printf("Please specify name of file!\n");
        exit(EXIT_FAILURE);
    }

	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

    // read input music file
	FILE *fp;
    fp = fopen(argv[1], "rb");
    if (!fp)
    {
        printf("open input file failed!\n");
        return -1;
    }
    // get info of file
    waveFormat fmt = readWaveHeader(fp);
    int size = fmt.data_size;
    
    // allocate host memory for input and output data
    signal = (element *)malloc(size * sizeof(element));
    // get data of input file
    fseek(fp, 44L, SEEK_SET);
    fread(signal, sizeof(short), size, fp);
    // close file stream
    if (fp)
    {
        fclose(fp);
        fp = NULL;
    }

    // allocate host memory for output data
	result = (element *)malloc(size * sizeof(element));

    // execute median filter and time it
	hipEventRecord(start, 0);
	medianfilter(signal, result, size);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("%.3lf ms\n", elapsedTime);

    // save output data
	fp = fopen("audios/gpu_v2_rst.wav", "wb+");

	if (fp == NULL)
        printf("Open output file failed!\n");

    writeWaveHeader(fmt, fp);
    fseek(fp, 44L, SEEK_SET);
    fwrite(result, sizeof(short), size, fp);
    
    // close file stream
    if (fp)
    {
	    fclose(fp);
        fp = NULL;
    }

    // free host memory
    free(signal);
    free(result);

	return 0;
}
